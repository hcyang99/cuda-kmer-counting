#include "hip/hip_runtime.h"
#include "Gerbil/Gerbil.cuh"
#include "benchmarks/HashtableWalker.cuh"
#include "utils/Fasta.cuh"
#include "utils/FailureZone.cuh"
#include "benchmarks/OccurenceArrayWalker.cuh"
#include <thrust/sort.h>
#include <thrust/scan.h>
#include <thrust/execution_policy.h>
#include <chrono>
using namespace std;

__global__
void init_failure_zone_kernel(FailureZone<Compressed128Mer>* location, Compressed128Mer* buffer, size_t size)
{
    if (utils::global_thread_id() == 0)
    {
        new (location) FailureZone<Compressed128Mer>(buffer, size);
    }
}

__global__ 
void gerbil_kernel(uint32_t* ref, uint32_t* d, uint32_t n, JobQueue* j, uint32_t max_trials, FailureZone<Compressed128Mer>* f_zone)
{
    __shared__ char s[sizeof(Gerbil)];
    int tx = threadIdx.x;

    Gerbil* h = (Gerbil*) s;
    if (tx == 0)
    {
        new (h) Gerbil(ref, d, n, j, max_trials, f_zone);
    }
    __syncthreads();

    h->run();
}

__host__ 
FailureZone<Compressed128Mer>* get_failure_zone(size_t size)
{
    Compressed128Mer* buffer; 
    FailureZone<Compressed128Mer>* location;
    CUDA_CHECK_ERROR(hipMalloc(&buffer, size * sizeof(Compressed128Mer)));
    CUDA_CHECK_ERROR(hipMalloc(&location, sizeof(FailureZone<Compressed128Mer>)));
    init_failure_zone_kernel<<<1, 32>>>(location, buffer, size);
    
    CUDA_CHECK_ERROR(hipDeviceSynchronize());

    return location;
}

__host__ 
uint32_t* gerbil(uint32_t* ref, size_t n_bps, size_t& n_buckets_out, uint32_t max_trials, 
    FailureZone<Compressed128Mer>*& f_zone_out, int64_t& us_out)
{
    size_t n_jobs = n_bps - 127;
    JobQueue* j = new_job(n_jobs, utils::BatchSize::get());
    size_t num_buckets = n_jobs * utils::OAFactor::get() / 14;

    uint32_t* d_buf;
    // cerr << "Allocating " << num_buckets << " buckets" << endl;
    CUDA_CHECK_ERROR(hipMalloc(&d_buf, num_buckets * 512));
    CUDA_CHECK_ERROR(hipMemset(d_buf, 0, num_buckets * 512));

    CUDA_CHECK_ERROR(hipDeviceSynchronize());
    auto t1 = chrono::high_resolution_clock::now();

    auto f_zone = get_failure_zone(n_jobs);

    gerbil_kernel<<<utils::GridSize::get(), utils::BlockSize::get()>>>
        (ref, d_buf, num_buckets, j, max_trials, f_zone);
    
    CUDA_CHECK_ERROR(hipDeviceSynchronize());
    auto t2 = chrono::high_resolution_clock::now();
    us_out = chrono::duration_cast<chrono::microseconds>(t2 - t1).count();

    n_buckets_out = num_buckets;
    f_zone_out = f_zone;
    return d_buf;
}

__global__ 
void kmer_diff_scan(Compressed128Mer* kmers, uint32_t* out, size_t size)
{
    int tx = utils::global_thread_id();
    if (tx == 0)
        out[tx] = 1;
    else if (tx < size)
    {
        out[tx] = kmers[tx - 1] == kmers[tx];
    }
}

struct KmerLess
{
    __device__ __host__ KmerLess() {}

    __device__ __host__ bool operator()(const Compressed128Mer& lhs, const Compressed128Mer& rhs) const 
    {
        for (int i = 0; i < 8; ++i)
        {
            if (lhs.u32[i] < rhs.u32[i])
                return true;
            else if (lhs.u32[i] > rhs.u32[i])
                return false;
        }
        return false;
    }
};

__global__ 
void symbol_compaction_kernel(uint32_t* diff_array, uint32_t* symbol_array, size_t size, uint32_t* out)
{
    int tx = utils::global_thread_id();
    if (tx == 0)
    {
        out[0] = 0;
    }
    if (tx < size && diff_array[tx])
    {
        out[symbol_array[tx]] = tx;
    }
}

__global__ 
void decompact_occurences_kernel(uint32_t* c, uint32_t* out, size_t size)
{
    int tx = utils::global_thread_id();
    if (tx < size)
        out[tx] = c[tx + 1] - c[tx];
}

__host__ 
void process_failure_zone(FailureZone<Compressed128Mer>* f_zone, Stats* stats, uint32_t* out)
{
    Compressed128Mer* f_zone_ptrs[3];
    CUDA_CHECK_ERROR(hipMemcpy(f_zone_ptrs, f_zone, 24, hipMemcpyDeviceToHost));

    Compressed128Mer* buffer = f_zone_ptrs[0];
    Compressed128Mer* end = f_zone_ptrs[2];
    size_t size = end - buffer;

    cout << "Walking through " << size << " failed entries\n";

    // Step 1: sort
    thrust::sort(thrust::device, buffer, end, KmerLess());

    // Step 2: locate diffs
    uint32_t* diff_array;
    CUDA_CHECK_ERROR(hipMalloc(&diff_array, size * sizeof(uint32_t)));
    kmer_diff_scan<<<size / utils::BlockSize::get() + 1, utils::BlockSize::get()>>>
        (buffer, diff_array, size);
    CUDA_CHECK_ERROR(hipFree(buffer));

    // Step 3: prefix scan on diff array
    uint32_t* symbol_array;
    CUDA_CHECK_ERROR(hipMalloc(&symbol_array, size * sizeof(uint32_t)));
    thrust::inclusive_scan(thrust::device, diff_array, diff_array + size, symbol_array);

    // Step 4: stream compaction (result is prefix scan of array of occurences of each kmer)
    uint32_t* occurences_compacted;
    uint32_t num_keys;
    CUDA_CHECK_ERROR(hipMemcpy(&num_keys, symbol_array + (size - 1), sizeof(uint32_t), hipMemcpyDeviceToHost));
    CUDA_CHECK_ERROR(hipMalloc(&occurences_compacted, (num_keys + 1) * sizeof(uint32_t)));
    symbol_compaction_kernel<<<size / utils::BlockSize::get() + 1, utils::BlockSize::get()>>>
        (diff_array, symbol_array, size, occurences_compacted);

    CUDA_CHECK_ERROR(hipFree(diff_array));
    CUDA_CHECK_ERROR(hipFree(symbol_array));

    // Step 5: map, output is array of occurences of each kmer
    uint32_t* occurence_array;
    CUDA_CHECK_ERROR(hipMalloc(&occurence_array, num_keys * sizeof(uint32_t)));
    decompact_occurences_kernel<<<num_keys / utils::BlockSize::get() + 1, utils::BlockSize::get()>>>
        (occurences_compacted, occurence_array, num_keys);

    CUDA_CHECK_ERROR(hipFree(occurences_compacted));

    // Step 6: occurence array walker
    occurence_table_walk(occurence_array, num_keys, stats, out);
}

int main(int argc, char** argv)
{
    if (argc != 4)
    {
        cerr << "Usage: " << argv[0] << " <path-to-fasta> <oa-factor> <max-trials>" << endl;
        exit(1);
    }

    string path(argv[1]);
    float oa_factor = strtof(argv[2], nullptr);
    int max_trials = strtol(argv[3], nullptr, 10);

    utils::OAFactor::set(oa_factor);

    uint32_t result[128];
    Fasta fasta(path);
    // cerr << "Reading " << fasta.size() << " base pairs" << endl;

    int64_t us_run, us_walk;

    uint32_t* d_ref = fasta.toGpuCompressed();
    size_t num_buckets;
    FailureZone<Compressed128Mer>* f_zone;
    uint32_t* d_table_buf = gerbil(d_ref, fasta.size(), num_buckets, max_trials, f_zone, us_run);

    // cerr << "Walking through " << num_buckets << " buckets: " << d_table_buf << endl;
    cout << "gerbil (" << oa_factor << ", " << max_trials << ") Insertion Time: " << us_run << " us\n";
    
    auto t1 = chrono::high_resolution_clock::now();
    Stats* stats = hashtable_walk(d_table_buf, num_buckets);
    process_failure_zone(f_zone, stats, result);
    auto t2 = chrono::high_resolution_clock::now();
    us_walk = chrono::duration_cast<chrono::microseconds>(t2 - t1).count();

    cout << "gerbil (" << oa_factor << ", " << max_trials << ") Walk Time: " << us_walk << " us\n";
}