#include "hip/hip_runtime.h"
#include "Fasta.cuh"
#include "utils.cuh"
#include <iostream>
#include <fstream>
#include <stdexcept>
using namespace std;

__host__ 
Fasta::Fasta(string path)
{
    ifstream f(path);
    if (!f)
        throw runtime_error("Cannot open file");
    
    string line;
    do 
    {
        getline(f, line);
        if (line.size() != 0 && line[0] != '>' && line[0] != ';')
        {
            this->buffer += line;
        }
    } 
    while(!f.eof() && !(line.size() != 0 && line[0] == '>'));
    f.close();

    this->sz = buffer.size();

    int remainder = this->buffer.size() % 16;
    int pad = remainder == 0 ? 0 : 16 - remainder;

    // pad with 'A' for compressing
    this->buffer += string(pad, 'A');
}

using utils::byte_32;
using utils::byte_4;

__device__ __forceinline__
byte_4 compress(const byte_32& in)
{
    byte_4 result = byte_4();
    
    for (uint32_t i = 0; i < 16; ++i)
    {
        uint8_t curr;
        switch (in.c[i])
        {
            case 'A': curr = 0; break;
            case 'C': curr = 1; break;
            case 'G': curr = 2; break;
            default: curr = 3; break;
        }
        uint32_t byte_idx = i / 4;
        uint32_t shift = (3 - i % 4) * 2;
        result.c[byte_idx] |= curr << shift;
    }

    return result;
}

/**
 * @brief Compress input ACGT string each to 2-bit
 * @param dst Destination buffer
 * @param src Source string buffer
 */
__global__
void compressKernel(uint32_t* dst, uint32_t* src)
{
    char* src_char = reinterpret_cast<char*>(src);

    int tx = utils::global_thread_id();

    byte_32 in;
    for (uint32_t i = 0; i < 4; ++i)
    {
        in.u32[i] = src[tx * 4 + i];
    }
    byte_4 result = compress(in);
    dst[tx] = result.u32;
}

__host__
uint32_t* Fasta::toGpuCompressed()
{
    uint32_t* d_buf;
    uint32_t* d_compressed;
    CUDA_CHECK_ERROR(hipMalloc(&d_buf, this->buffer.size()));
    CUDA_CHECK_ERROR(hipMalloc(&d_compressed, this->buffer.size() / 4));
    CUDA_CHECK_ERROR(hipMemcpy(d_buf, &this->buffer[0], this->buffer.size(), hipMemcpyHostToDevice));

    compressKernel<<<this->buffer.size() / 16, utils::blockSize()>>>
    (d_compressed, d_buf);

    hipFree(d_buf);
    this->buffer.clear();

    return d_compressed;
}