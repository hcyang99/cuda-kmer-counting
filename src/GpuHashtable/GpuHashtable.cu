#include "hip/hip_runtime.h"
#include "GpuHashtable.cuh"

__device__
GpuHashtable::ProbeStatus GpuHashtable::simd_probe(uint32_t* data, const Compressed128Mer& key, uint32_t* match_status, uint32_t* empty_status)
{
    int tx = threadIdx.x;
    int subwarp_idx = tx / 14;  // 14 (32B key, 4B value) pairs
    int sub_tx = tx % 9;        // 8 threads process the key, 1 thread processes the value
    if (tx < 126)               // let the last 2 threads rest
        if (sub_tx == 0)   
        {
            match_status[subwarp_idx] = 1;  // initialize as matched
        }
    __syncthreads();

    if (tx < 126)
        if (sub_tx != 8)
            if (data[tx] != key.u32[sub_tx])
            {
                match_status[subwarp_idx] = 0;  // set to 0 if not matching
            }
        else
        {
            empty_status[subwarp_idx] = data[tx] == 0;
        }
    __syncthreads();

    if (tx >= 0 && tx < 14)
    {
        match_status[14] = __ballot_sync(__activemask(), match_status[tx]);
    }
    else if (tx >= 32 && tx < 32 + 14)
    {
        empty_status[14] = __ballot_sync(__activemask(), empty_status[tx - 32]);
    }
    __syncthreads();


    int match_sub_block = __ffs(match_status[14]) - 1;
    int empty_sub_block = __ffs(empty_status[14]) - 1;
    if (tx == 0 && match_sub_block >= 0)
    {
        // match found, incrementing
        // deletions from hashtables not implemented
        int offset = 9 * match_sub_block + 8;
        atomicAdd(data + offset, 1UL);
    }
    if (match_sub_block >= 0)
        return ProbeStatus::SUCCEESS;
    if (empty_sub_block >= 0)
        return ProbeStatus::INSERT;
    return ProbeStatus::PROBE;
}